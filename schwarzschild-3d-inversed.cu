#include "hip/hip_runtime.h"
// Compile with:
// nvcc -O3 -o schwarzschild-3d-inversed schwarzschild-3d-inversed.cu -lGL -lGLU -lglut

#include <GL/glut.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <ctime>

const int WIDTH = 1920, HEIGHT = 1080;
const int NUM_PARTICLES = 9999999;
const float GM = 5.0f;
const float DT = 0.02f;

struct Particle {
    float r, theta, phi;
    float dr, dtheta, dphi;
    float color;
};

Particle* d_particles;
Particle* h_particles = new Particle[NUM_PARTICLES];
hiprandState* d_states;

// Camera
float camX = 0, camY = 0, camZ = 200;
float camYaw = 0, camPitch = 0;
bool keys[256] = { false };
int lastX, lastY;
bool dragging = false;

// Host version of spherical to cartesian
float3 spherical_to_cartesian_host(float r, float theta, float phi) {
    return {
        r * sinf(phi) * cosf(theta),
        r * cosf(phi),
        r * sinf(phi) * sinf(theta)
    };
}

__global__ void init_particles(Particle* p, hiprandState* states, int seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NUM_PARTICLES) return;

    hiprand_init(seed, i, 0, &states[i]);

    float r0 = 2.0f + hiprand_uniform(&states[i]) * 10.0f;
    float theta = hiprand_uniform(&states[i]) * 2 * M_PI;
    float phi = hiprand_uniform(&states[i]) * M_PI;

    p[i] = {
        r0, theta, phi,
        0.0f, 0.002f, 0.002f,
        1.0f
    };
}

__global__ void update_geodesics(Particle* p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NUM_PARTICLES) return;

    float r = p[i].r;
    float f = 1.0f + 2.0f * GM / r;
    float acc = -GM / (r * r);  // repulsive

    p[i].dr += acc * DT;
    p[i].r += p[i].dr * f * DT;
    p[i].theta += p[i].dtheta;
    p[i].phi += p[i].dphi;

    if (p[i].r > 100.0f) p[i].color = 0.2f;
}

void update_camera() {
    float speed = 2.0f;
    float yawRad = camYaw * M_PI / 180.0f;
    float pitchRad = camPitch * M_PI / 180.0f;

    float dx = cosf(yawRad);
    float dz = sinf(yawRad);

    if (keys['w']) { camX += dx * speed; camZ += dz * speed; }
    if (keys['s']) { camX -= dx * speed; camZ -= dz * speed; }
    if (keys['a']) { camX += dz * speed; camZ -= dx * speed; }
    if (keys['d']) { camX -= dz * speed; camZ += dx * speed; }
    if (keys['q']) { camY -= speed; }
    if (keys['e']) { camY += speed; }
}

void display() {
    update_camera();
    update_geodesics<<<(NUM_PARTICLES + 255) / 256, 256>>>(d_particles);
    hipMemcpy(h_particles, d_particles, NUM_PARTICLES * sizeof(Particle), hipMemcpyDeviceToHost);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glLoadIdentity();

    float yawRad = camYaw * M_PI / 180.0f;
    float pitchRad = camPitch * M_PI / 180.0f;
    float lx = cosf(pitchRad) * cosf(yawRad);
    float ly = sinf(pitchRad);
    float lz = cosf(pitchRad) * sinf(yawRad);

    gluLookAt(camX, camY, camZ, camX + lx, camY + ly, camZ + lz, 0, 1, 0);

    glBegin(GL_POINTS);
    for (int i = 0; i < NUM_PARTICLES; ++i) {
        Particle& p = h_particles[i];
        float3 pos = spherical_to_cartesian_host(p.r, p.theta, p.phi);
        glColor3f(p.color, p.color, p.color);
        glVertex3f(pos.x, pos.y, pos.z);
    }
    glEnd();

    glutSwapBuffers();
}

void idle() {
    glutPostRedisplay();
}

void keyDown(unsigned char key, int, int) {
    keys[key] = true;
}
void keyUp(unsigned char key, int, int) {
    keys[key] = false;
}

void mouse(int button, int state, int x, int y) {
    if (button == GLUT_LEFT_BUTTON) {
        dragging = (state == GLUT_DOWN);
        lastX = x;
        lastY = y;
    }
}

void motion(int x, int y) {
    if (dragging) {
        camYaw += (x - lastX) * 0.3f;
        camPitch -= (y - lastY) * 0.3f;
        if (camPitch > 89.0f) camPitch = 89.0f;
        if (camPitch < -89.0f) camPitch = -89.0f;
        lastX = x;
        lastY = y;
    }
}

void init_cuda() {
    hipMalloc(&d_particles, NUM_PARTICLES * sizeof(Particle));
    hipMalloc(&d_states, NUM_PARTICLES * sizeof(hiprandState));
    init_particles<<<(NUM_PARTICLES + 255) / 256, 256>>>(d_particles, d_states, time(0));
    hipDeviceSynchronize();
}

int main(int argc, char** argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(WIDTH, HEIGHT);
    glutCreateWindow("Inverted Schwarzschild 3D - Free Camera");

    glEnable(GL_DEPTH_TEST);
    glClearColor(0, 0, 0, 1);
    glPointSize(1.0f);

    glMatrixMode(GL_PROJECTION);
    gluPerspective(60.0, (float)WIDTH / HEIGHT, 0.1, 2000.0);
    glMatrixMode(GL_MODELVIEW);

    glutDisplayFunc(display);
    glutIdleFunc(idle);
    glutKeyboardFunc(keyDown);
    glutKeyboardUpFunc(keyUp);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);

    init_cuda();
    glutMainLoop();
    return 0;
}
